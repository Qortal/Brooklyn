#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "color_conversion.h"

#include "stdio.h"

__host__ __device__ __forceinline__ int divUp(int total, int grain)
{
	return (total + grain - 1) / grain;
}

__global__ void RGB_to_jp(uchar4 *input, unsigned char *yuv_luma, unsigned char *yuv_cb, unsigned char *yuv_cr, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x >= width || y>=height) return;

	uchar4 px = input[y * width + x];
	int Y = ( (  66 * px.x + 129 * px.y +  25 * px.z + 128) >> 8) +  16;
	int U = ( ( -38 * px.x -  74 * px.y + 112 * px.z + 128) >> 8) + 128;
	int V = ( ( 112 * px.x -  94 * px.y -  18 * px.z + 128) >> 8) + 128;
	
	yuv_luma[y * width + x] = Y;

	int pos = (y >> 1) * (width >> 1) + (x >> 1);
	yuv_cr[pos] = U;
	yuv_cb[pos] = V;
}

bool RGB_to_YV12(int width, int height, void *pPixels, void* yuv_luma, void* yuv_cb, void* yuv_cr)
{
	hipError_t cudaStatus;

	const dim3 block(32, 8);
	const dim3 grid(divUp(width, block.x), divUp(height, block.y));

	unsigned char *yuv_luma_device;
	hipMalloc(&yuv_luma_device, width *height * sizeof(unsigned char));

	unsigned char *yuv_cb_device;
	hipMalloc(&yuv_cb_device, width *height * sizeof(unsigned char) / 4);

	unsigned char *yuv_cr_device;
	hipMalloc(&yuv_cr_device, width *height * sizeof(unsigned char) / 4);
   

	// Copy input vectors from host memory to GPU buffers.
	uchar4 *dev_pPixels;
	cudaStatus = hipMalloc((void**)&dev_pPixels, width *height * sizeof(uchar4));
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return false;
    }

	cudaStatus = hipMemcpy(dev_pPixels, pPixels, width *height * sizeof(uchar4), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 1 failed!");
        return false;
    }

	RGB_to_jp<<< grid, block >>>(dev_pPixels, yuv_luma_device, yuv_cb_device, yuv_cr_device, width, height);
   
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "RGB_to_jp failed!");
        return false;
    }
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!");
        return false;
    }

	cudaStatus = hipMemcpy(yuv_luma, yuv_luma_device, width *height * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 2 failed!");
        return false;
    }
	cudaStatus = hipMemcpy(yuv_cb, yuv_cb_device, width *height * sizeof(unsigned char) / 4, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 2 failed!");
        return false;
    }
	cudaStatus = hipMemcpy(yuv_cr, yuv_cr_device, width *height * sizeof(unsigned char) / 4, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 2 failed!");
        return false;
    }

	hipFree(yuv_luma_device);
	hipFree(yuv_cb_device);
	hipFree(yuv_cr_device);
	hipFree(dev_pPixels);

    return true;
}